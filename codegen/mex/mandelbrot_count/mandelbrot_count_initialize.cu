//
//  mandelbrot_count_initialize.cu
//
//  Code generation for function 'mandelbrot_count_initialize'
//


// Include files
#include "mandelbrot_count_initialize.h"
#include "_coder_mandelbrot_count_mex.h"
#include "mandelbrot_count.h"
#include "mandelbrot_count_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void mandelbrot_count_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (mandelbrot_count_initialize.cu)
