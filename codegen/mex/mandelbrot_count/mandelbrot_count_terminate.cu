//
//  mandelbrot_count_terminate.cu
//
//  Code generation for function 'mandelbrot_count_terminate'
//


// Include files
#include "mandelbrot_count_terminate.h"
#include "_coder_mandelbrot_count_mex.h"
#include "mandelbrot_count.h"
#include "mandelbrot_count_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void mandelbrot_count_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void mandelbrot_count_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (mandelbrot_count_terminate.cu)
